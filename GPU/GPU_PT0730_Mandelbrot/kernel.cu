﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

void pointApproximation(float* realPart, float* imagPart, int* maxIter, int* approximation);

int main()
{
    float realPoint = 0.2;
    float imagPoint = 0.2;
    int max = 100;
    int appro = 0;
    pointApproximation(&realPoint, &imagPoint, &max, &appro);
    cout << appro;

    return 0;
}

void pointApproximation(float* realPart, float* imagPart, int *maxIter, int *approximation)
{
    int i = 0;
    float zReal = 0;
    float zImag = 0;
    float zTempReal = 0;
    float zTempImag = 0;

    while (i < *maxIter && (zReal * zReal + zImag * zImag < 4))
    {
        zTempReal = zReal * zReal - zImag * zImag;
        zTempImag = 2 * zReal * zImag;

        zReal = zTempReal + *realPart;
        zImag = zTempImag + *imagPart;


        i++;
    }

    *approximation = i;
}