#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <time.h>
#include <windows.h>

using namespace std;


double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
    LARGE_INTEGER li;
    if (!QueryPerformanceFrequency(&li))
        cout << "QueryPerformanceFrequency failed!\n";

    PCFreq = double(li.QuadPart) / 1000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
}
double GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return double(li.QuadPart - CounterStart) / PCFreq;
}



__device__ void pointApproximationCount(float* realPart, float* imagPart, int* maxIter, int* approximation)
{
    *approximation = 0;
    int i = 0;
    float zReal = 0;
    float zImag = 0;
    float zTempReal = 0;
    float zTempImag = 0;

    while (i < *maxIter && (zReal * zReal + zImag * zImag < 4))
    {
        zTempReal = zReal * zReal - zImag * zImag;
        zTempImag = 2 * zReal * zImag;

        zReal = zTempReal + *realPart;
        zImag = zTempImag + *imagPart;


        i++;
    }

    *approximation = i;
    printf("%d", *approximation);
}

__device__ void traverseCount(float* startX, float* startY, float endX, float endY, float* step, int* maxIter, int* approximation, float* width)
{
    int i = 0;
    float curX, curY;
    curY = *startY;
    while (curY < endY) {
        int j = 0;
        curX = *startX;
        while (curX < endX) {
            pointApproximation(&curX, &curY, maxIter, approximation); //+ (i * (int)(*width / *step + 0.5)) + j++);
            curX += *step;
            j++;
        }
        curY += *step;
        i++;
    }
}

__global__ void FuncCount(float* startX, float* startY, int* appro, float* step, int* maxIter, float* width, int* numberThreads, int size, float ratio)
{
    int i = threadIdx.x;
    if (i < *numberThreads - 1)
        traverse(startX + i, startY + i, 2.0f, startY[i + 1], step, maxIter, appro, width);
    else
        traverse(startX + i, startY + i, 2.0f, 2.0f, step, maxIter, appro, width);
}


__device__ void traverse(float* startX, float* startY, float endX, float endY, float* step, int* maxIter, int* approximation, float* width)
{
    int i = 0;
    float curX, curY;
    curY = *startY;
    while (curY < endY) {
        int j = 0;
        curX = *startX;
        while (curX < endX) {
            pointApproximation(&curX, &curY, maxIter, approximation + (i * (int)(*width / *step + 0.5)) + j++);
            curX += *step;
        }
        curY += *step;
        i++;
    }
}

__global__ void Func(float* startX, float* startY, int* appro, float* step, int* maxIter, float* width, int* numberThreads, int size, float ratio)
{
    int i = threadIdx.x;
    if (i < *numberThreads - 1)
        traverse(startX + i, startY + i, 2.0f, startY[i + 1], step, maxIter, appro + (int)(size * size * i / ratio + 0.5), width);
    else
        traverse(startX + i, startY + i, 2.0f, 2.0f, step, maxIter, appro + (int)(size * size * i / ratio + 0.5), width);
}

int main()
{
    int max = 0;
    float step = 0;
    int numberThreads = 0;

    cout << "Prosze podac dokladnosc liczby zmiennoprzecinkowej: ";
    cin >> step;
    cout << "\nProsze podac maksymalna liczbe powtorzen funkcji sprawdzajacej przynaleznosc punktu do zbioru: ";
    cin >> max;
    //cout << "\nProsze podac liczbe watkow: ";
    //cin >> numberThreads;

    int deviceCount, device;
    int gpuDeviceCount = 0;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess)
        deviceCount = 0;
    for (device = 0; device < deviceCount; ++device) 
    {
        hipGetDeviceProperties(&properties, device);
        //9999 means emulation only
        if (properties.major != 9999)
            if (device == 0)
            {
                numberThreads = properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor;
                cout << numberThreads << endl;
            }
    }

    double ms = 0;
    StartCounter();

    float width = 4.0;
    int size = (int)((width / step) + 0.5);

    if (numberThreads > size)
        numberThreads = size;   //clamp(0,size)

    float* realPoints;
    realPoints = (float*)malloc(sizeof(float) * numberThreads);

    float* imagPoints;
    imagPoints = (float*)malloc(sizeof(float) * numberThreads);



    int most_of_rectangles_height = size / numberThreads;

    //Most of the rectangles size ratio
    float most_of_rectangles_size_ratio = size / (float)most_of_rectangles_height;


    for (int i = 0; i < numberThreads; i++)
    {
        realPoints[i] = -2.0;
        imagPoints[i] = -2.0 + width * i / most_of_rectangles_size_ratio;
    }

    float* realPoints_c, * imagPoints_c;
    int* approximations_c;
    float* width_c;
    int* max_c;
    float* step_c;
    int* numberThreads_c;

    hipMalloc((void**)&realPoints_c, sizeof(float) * numberThreads);
    hipMalloc((void**)&imagPoints_c, sizeof(float) * numberThreads);


    hipMalloc((void**)&width_c, sizeof(float));
    hipMalloc((void**)&max_c, sizeof(int));
    hipMalloc((void**)&step_c, sizeof(float));
    hipMalloc((void**)&numberThreads_c, sizeof(int));

    hipMemcpy(realPoints_c, realPoints, sizeof(float) * numberThreads, hipMemcpyHostToDevice);
    hipMemcpy(imagPoints_c, imagPoints, sizeof(float) * numberThreads, hipMemcpyHostToDevice);
    hipMemcpy(width_c, &width, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(max_c, &max, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(step_c, &step, sizeof(float), hipMemcpyHostToDevice);

    int* approximations;
    if (size <= 0)
    {
        approximations = (int*)malloc(sizeof(int) * (size + 1) * (size + 1));
        hipMalloc((void**)&approximations_c, sizeof(int) * (size + 1) * (size + 1));
        hipMemcpy(approximations_c, approximations, sizeof(int) * (size + 1) * (size + 1), hipMemcpyHostToDevice);

        Func << <1, numberThreads >> > (realPoints_c, imagPoints_c, approximations_c, step_c, max_c, width_c, numberThreads_c, size, most_of_rectangles_size_ratio);
    }
    else
    {
        approximations = (int*)malloc(sizeof(int));
        hipMalloc((void**)&approximations_c_c, sizeof(int));
        hipMemcpy(approximations_c, &approximations, sizeof(int), hipMemcpyHostToDevice);

        FuncCount <<<1,numberThreads>>>(realPoints_c, imagPoints_c, approximations_c, step_c, max_c, width_c, numberThreads_c, size, most_of_rectangles_size_ratio)
    }



    ms += GetCounter();

    if (size <= 0)
    {
        hipMemcpy(approximations, approximations_c, sizeof(int) * (size + 1) * (size + 1), hipMemcpyDeviceToHost);


        for (int i = 0; i < (size); i++) {
            for (int j = 0; j < size; j++) {
                cout << setw(2) << approximations[i * size + j];
            }
            cout << endl;
        }

        cout << "\r\n\r\n" << ms << "ms" << "\r\n";

        //std::fstream file("mandelbrot.pgm", std::fstream::out);
        //file << "P2\n" << size << " " << size << "\n" << max << "\n";
        //std::string line, value;

        //line = "";
        //for (int i = 0; i < size * size; i++)
        //{
        //    value = to_string(approximations[(int)(i)]);
        //    if (line.length() + value.length() > 69)
        //    {
        //        file << line << "\n";
        //        line = "";
        //    }
        //    line += value + " ";
        //}

        //file << line;

        //file.close();
    }
    else
        cout << "\r\n\r\n" << ms << "ms" << "\r\n";

    return 0;
}


