#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__device__ void pointApproximation(float* realPart, float* imagPart, int* maxIter, int* approximation)
{
    *approximation = 0;
    int i = 0;
    float zReal = 0;
    float zImag = 0;
    float zTempReal = 0;
    float zTempImag = 0;

    while (i < *maxIter && (zReal * zReal + zImag * zImag < 4))
    {
        zTempReal = zReal * zReal - zImag * zImag;
        zTempImag = 2 * zReal * zImag;

        zReal = zTempReal + *realPart;
        zImag = zTempImag + *imagPart;


        i++;
    }

    *approximation = i;
}

__device__ void traverse(float* startX, float* startY, float endX, float endY, float* step, int* maxIter, int* approximation, float* width)
{
    int i = 0;
    float curX, curY;
    curY = *startY;
    while (curY < endY) {
        int j = 0;
        curX = *startX;
        while (curX < endX) {
            pointApproximation(&curX, &curY, maxIter, approximation + (i * (int)(*width / *step + 0.5)) + j++);
            curX += *step;
        }
        curY += *step;
        i++;
    }
}

__global__ void Func(float* startX, float* startY, int* appro, float* step, int* maxIter, float* width, int* numberThreads)
{
    int i = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * 8 + threadIdx.x + threadIdx.y * 8;
    traverse(startX + i, startY + i, (startX[i] + *width), (startY[i] + *width), step, maxIter, appro + (8*((i/8)*4))+2*(i%8), width);
}

int main()
{
    int max = 0;
    float step = 0;

    cout << "Prosze podac dokladnosc liczby zmiennoprzecinkowej: ";
    cin >> step;
    cout << "\nProsze podac maksymalna liczbe powtorzen funkcji sprawdzajacej przynaleznosc punktu do zbioru: ";
    cin >> max;

    int numberThreads = 8;
    float width = 4.0 / numberThreads;

    float* realPoints;
    realPoints = (float*)malloc(sizeof(float) * numberThreads*numberThreads);

    float* imagPoints;
    imagPoints = (float*)malloc(sizeof(float) * numberThreads*numberThreads);

    int* approximations;
    approximations = (int*)malloc(sizeof(int) * (width/step)* (width / step)* numberThreads * numberThreads);

    for (int i = 0; i < (width / step) * (width / step) * numberThreads * numberThreads; i++) {
        approximations[i] = -2;
    }

    for (int i = 0; i < numberThreads; i++)
    {
        for (int j = 0; j < numberThreads; j++)
        {
            realPoints[i * numberThreads + j] = -2.0 + width * i;
            imagPoints[i * numberThreads + j] = -2.0 + width * j;
        }
    }

    float* realPoints_c, *imagPoints_c;
    int* approximations_c;
    float* width_c;
    int* max_c;
    float* step_c;
    int* numberThreads_c;

    hipMalloc((void**)&realPoints_c, sizeof(float) * numberThreads*numberThreads);
    hipMalloc((void**)&imagPoints_c, sizeof(float) * numberThreads * numberThreads);
    hipMalloc((void**)&approximations_c, sizeof(int) * (width / step) * (width / step) * numberThreads * numberThreads);
    hipMalloc((void**)&width_c, sizeof(float));
    hipMalloc((void**)&max_c, sizeof(int));
    hipMalloc((void**)&step_c, sizeof(float));
    hipMalloc((void**)&numberThreads_c, sizeof(int));

    hipMemcpy(realPoints_c, realPoints, sizeof(float) * numberThreads*numberThreads, hipMemcpyHostToDevice);
    hipMemcpy(imagPoints_c, imagPoints, sizeof(float) * numberThreads * numberThreads, hipMemcpyHostToDevice);
    hipMemcpy(width_c, &width, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(max_c, &max, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(step_c, &step, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(approximations_c, approximations, sizeof(int) * (width / step) * (width / step) * numberThreads * numberThreads, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(4, 4);
    dim3 numBlocks(numberThreads / threadsPerBlock.x, numberThreads / threadsPerBlock.y);

    Func << <numBlocks, threadsPerBlock >> > (realPoints_c, imagPoints_c, approximations_c, step_c, max_c, width_c, numberThreads_c);

    hipMemcpy(approximations, approximations_c, sizeof(int) * (width / step) * (width / step) * numberThreads * numberThreads, hipMemcpyDeviceToHost);

    for (int i = 0; i < (width / step) * (width / step) * numberThreads * numberThreads; i++)
    {
        if (i % (int)((width / step) * numberThreads + 0.5) == 0)
            cout << endl;
        if(approximations[i]>=0)
            cout << approximations[i] << "\t";
        else
            cout << "  \t";
    }
    return 0;
}


